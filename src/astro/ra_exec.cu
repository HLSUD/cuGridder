#include "hip/hip_runtime.h"
/*
INVERSE: type 1
FORWARD: type 2
*/
#include <iostream>
#include <iomanip>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <complex>
#include <hipfft/hipfft.h>

#include "curafft_plan.h"
#include "conv_interp_invoker.h"
#include "deconv.h"
#include "precomp.h"
#include "ragridder_plan.h"
#include "ra_exec.h"
#include "cuft.h"
#include "utils.h"
#include <fstream>
using namespace std;


__global__ void div_n_lm(CUCPX *fk, PCS xpixelsize, PCS ypixelsize, int N1, int N2)
{
    int idx;
    PCS n_lm;
    int row, col;
    for (idx = blockDim.x * blockIdx.x + threadIdx.x; idx < N1 * N2; idx += gridDim.x * blockDim.x)
    {
        row = idx / N1;
        col = idx % N1;
        n_lm = sqrt(1.0 - pow((row - N2 / 2) * xpixelsize, 2) - pow((col - N1 / 2) * ypixelsize, 2));

        fk[idx].x /= n_lm;
        fk[idx].y /= n_lm;
        
    }
}

int cura_rscaling(CURAFFT_PLAN *plan, ragridder_plan *gridder_plan)
{
    //  * 1/n
    int N1 = gridder_plan->width;
    int N2 = gridder_plan->height;
    int N = N1 * N2;
    int blocksize = 256;
    int gridsize = (N - 1) / blocksize + 1;

    div_n_lm<<<gridsize, blocksize>>>(plan->fk, gridder_plan->pixelsize_x, gridder_plan->pixelsize_y, N1, N2);
    checkCudaErrors(hipDeviceSynchronize());

    return 0;
}


//
__global__ void shift_corr(CUCPX *d_c, PCS *d_w, PCS i_center, PCS o_center, PCS gamma, int nrow, int flag){
        int idx;
        for(idx=threadIdx.x + blockDim.x*blockIdx.x; idx<nrow; idx+=gridDim.x*blockDim.x){
                PCS phase = (d_w[idx]*gamma+i_center)*o_center*flag;
                CUCPX temp;
                temp.x = d_c[idx].x * cos(phase) - d_c[idx].y * sin(phase);
                temp.y = d_c[idx].x * sin(phase) + d_c[idx].y * cos(phase);
                d_c[idx] = temp;
        }
}
int shift_corr_invoker(CUCPX *d_c, PCS *d_w, PCS i_center, PCS o_center, PCS gamma, int nrow, int flag){
    // 
    int blocksize = 512;
    shift_corr<<<(nrow-1)/blocksize+1,blocksize>>>(d_c,d_w,i_center,o_center,gamma,nrow,flag);
    checkCudaErrors(hipDeviceSynchronize());
    return 0;
}
int cura_fw(CURAFFT_PLAN *plan, ragridder_plan *gridder_plan){
    // /wgt
    if(gridder_plan->kv.weight!=NULL){
        PCS *d_wgt;
        int nrow = gridder_plan->nrow;
        checkCudaErrors(hipMalloc((void**)&d_wgt,sizeof(PCS)*nrow));
        checkCudaErrors(hipMemcpy(d_wgt,gridder_plan->kv.weight,sizeof(PCS)*nrow,hipMemcpyHostToDevice));

        matrix_elementwise_divide_invoker(plan->d_c,d_wgt,nrow);
        checkCudaErrors(hipFree(d_wgt)); // to save memory
   }
    int ier = 0;
    int flag = plan->iflag;
    PCS gamma = plan->ta.gamma[0];
    shift_corr_invoker(plan->d_c,plan->d_w,plan->ta.i_center[0],plan->ta.o_center[0],gamma,gridder_plan->nrow,flag);
    return ier;
}


int cura_prestage(CURAFFT_PLAN *plan, ragridder_plan *gridder_plan){
        int ier = 0;
        int nrow = gridder_plan -> nrow;
        int N1 = plan->ms;
        int N2 = plan->mt;
        if (plan->iflag==1){
                if(gridder_plan->kv.weight!=NULL&&plan->copts.direction==1){
                        PCS *d_wgt;
                        checkCudaErrors(hipMalloc((void**)&d_wgt,sizeof(PCS)*nrow));
                        checkCudaErrors(hipMemcpy(d_wgt,gridder_plan->kv.weight,sizeof(PCS)*nrow,hipMemcpyHostToDevice));

                        matrix_elementwise_multiply_invoker(plan->d_c,d_wgt,nrow);
                        checkCudaErrors(hipFree(d_wgt)); // to save memory
                }
                // u_j to u_j' x_k to x_k' c_j to c_j'
                checkCudaErrors(hipMalloc((void **)&plan->d_x, sizeof(PCS) * (N1 / 2 + 1) * (N2 / 2 + 1)));
                w_term_k_generation(plan->d_x, N1, N2, gridder_plan->pixelsize_x, gridder_plan->pixelsize_y);
                pre_stage_1_invoker(plan->ta.o_center,plan->d_w,NULL,NULL,plan->d_c,nrow,plan->iflag);
                pre_stage_2_invoker(plan->ta.i_center, plan->ta.o_center, plan->ta.gamma, plan->ta.h, plan->d_w, NULL, NULL, plan->d_x, NULL, NULL, plan->d_c, gridder_plan->nrow,(N1 / 2 + 1) * (N2 / 2 + 1), 1, 1);
                
        }
        else{
                
                // u_j to u_j' x_k to x_k' fk to fk'
                checkCudaErrors(hipMalloc((void **)&plan->d_x, sizeof(PCS) * (N1 / 2 + 1) * (N2 / 2 + 1)));
                w_term_k_generation(plan->d_x, N1, N2, gridder_plan->pixelsize_x, gridder_plan->pixelsize_y);
                // pre_stage_1_invoker(plan->ta.i_center[0],plan->d_x,plan->fk,N1,N2,gridder_plan->pixelsize_x, gridder_plan->pixelsize_y,plan->iflag);
                pre_stage_2_invoker(plan->ta.i_center, plan->ta.o_center, plan->ta.gamma, plan->ta.h, plan->d_w, NULL, NULL, plan->d_x, NULL, NULL, plan->d_c, gridder_plan->nrow,(N1 / 2 + 1) * (N2 / 2 + 1), 1, 1);
        }

        fourier_series_appro_invoker(plan->fwkerhalf1, plan->copts, plan->nf1 / 2 + 1, plan->opts.gpu_kerevalmeth);
        fourier_series_appro_invoker(plan->fwkerhalf2, plan->copts, plan->nf2 / 2 + 1, plan->opts.gpu_kerevalmeth);
        int w_term_method = 1;
        if (w_term_method)
        {
                // improved_ws
                checkCudaErrors(hipFree(plan->fwkerhalf3));
                checkCudaErrors(hipMalloc((void **)&plan->fwkerhalf3, sizeof(PCS) * (N1 / 2 + 1) * (N2 / 2 + 1)));
                fourier_series_appro_invoker(plan->fwkerhalf3, plan->d_x, plan->copts, (N1 / 2 + 1) * (N2 / 2 + 1), plan->opts.gpu_kerevalmeth); // correction with k, may be wrong, k will be free in this function
        }

        // the pirange issue!!!
        // bin mapping

        if(plan->mem_limit){
                gridder_plan->temp_station = (int *) malloc (sizeof(int)*(((plan->nf1-1)/plan->hivesize[0] + 1)*((plan->nf2-1)/plan->hivesize[1] + 1)+1));
                part_bin_mapping_pre(plan, gridder_plan->temp_station, plan->initial);
                checkCudaErrors(hipMalloc((void**)&plan->fw_temp, plan->nf1 * plan->nf2 * sizeof(CUCPX)));/// free somewhere | for dft
                checkCudaErrors(hipMemset(plan->fw_temp,0,plan->nf1 * plan->nf2 * sizeof(CUCPX)));
        } 
        else if(plan->opts.gpu_gridder_method!=0)bin_mapping(plan,gridder_plan->d_uvw); //currently just support 3d //uvw or u?
        // fw malloc
        unsigned long long int fw_size = plan->nf1;
        fw_size *= plan->nf2;
        fw_size *= plan->nf3;
        checkCudaErrors(hipMalloc((void**)&plan->fw, fw_size * sizeof(CUCPX)));
        checkCudaErrors(hipMemset(plan->fw, 0, fw_size * sizeof(CUCPX)));
        return ier;
}

// int cura_cscaling(CURAFFT_PLAN *plan, ragridder_plan *gridder_plan)
// {
//     int N = gridder_plan->nrow;
//     return 0;
// }

int exec_vis2dirty(CURAFFT_PLAN *plan, ragridder_plan *gridder_plan)
{
    /*
    Currently, just suitable for improved W stacking
    Two different execution flows
        Flow1: the data size is relatively small and memory is sufficent for whole conv
        Flow2: the data size is too large, the data is divided into parts 
    */
    int ier = 0;
    //printf("execute flow %d\n",plan->execute_flow);
    if (plan->execute_flow == 1)
    {
        /// curafft_conv workflow for enough memory
#ifdef DEBUG
        printf("plan info printing...\n");
        printf("nf (%d,%d,%d), upsampfac %lf\n", plan->nf1, plan->nf2, plan->nf3, plan->copts.upsampfac);
        printf("gridder_plan info printing...\n");
        printf("fov %lf, current channel %d, w_s_r %lf\n", gridder_plan->fov, gridder_plan->cur_channel, gridder_plan->w_s_r);
#endif
        // 0. pre-stage
        ier = cura_prestage(plan,gridder_plan);
        // 1. convlution
#ifdef TIME
        hipEvent_t start, stop;
        float milliseconds = 0;
        float totaltime = 0;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
#endif
        if(!plan->mem_limit){ier = curafft_conv(plan);

        // printf("conv result printing (first w plane)...\n");
        // ofstream myfile;
  	// // myfile.open ("result2.txt");
        // CPX *fw = (CPX *)malloc(sizeof(CPX) * plan->nf1 * plan->nf2 *24 );

        // hipMemcpy(fw, plan->fw +plan->nf1 * plan->nf2*24, sizeof(CUCPX) * plan->nf1 * plan->nf2 * 6, hipMemcpyDeviceToHost);
        // printf("%.6g\n",fw[107532].real());
        // for (int j = 0; j < plan->nf1 * plan->nf2*6; j++)
        // {
        //         // if(fw[j].real()!=0){printf("%d, %.6g\n",j,fw[j].real());break;}
        //         myfile<<fw[j].real()<<"\n";
        // }
        // myfile<<"\n";
        // myfile.close();
        // show_mem_usage();
        }
#ifdef TIME
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        totaltime += milliseconds;
        printf("[time  ] conv time:\t\t %.3g s\n", milliseconds / 1000);
#endif
       
#ifdef DEBUG
        printf("conv result printing (first w plane)...\n");
        CPX *fw = (CPX *)malloc(sizeof(CPX) * plan->nf1 * plan->nf2 );
        hipMemcpy(fw, plan->fw, sizeof(CUCPX) * plan->nf1 * plan->nf2, hipMemcpyDeviceToHost);
        PCS temp = 0;
        for (int i = 0; i < 200; i++)
        {
                temp += fw[i].real();
                printf("%.3g ", fw[i].real());
        }

#endif
        // printf("n1 n2 n3 M %d, %d, %d, %d\n",plan->nf1,plan->nf2,plan->nf3,plan->M);
        // 2. cufft
#ifdef TIME
        hipEventRecord(start);
#endif
        if(!plan->mem_limit){
                cufft_plan_setting(plan);
                cura_cufft(plan);
        }
        // int direction = plan->iflag;
        // // cautious, a batch of fft, bath size is num_w when memory is sufficent.
        
        // CUFFT_EXEC(plan->fftplan, plan->fw, plan->fw, direction); // sychronized or not
        // hipDeviceSynchronize();
#ifdef TIME
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        totaltime += milliseconds;
        printf("[time  ] cufft time:\t\t %.3g s\n", milliseconds / 1000);
#endif
#ifdef DEBUG
        printf("fft result printing (first w plane)...\n");
        //CPX *fw = (CPX *)malloc(sizeof(CPX)*plan->nf1*plan->nf2*plan->nf3);
        hipMemcpy(fw, plan->fw, sizeof(CUCPX) * plan->nf1 * plan->nf2 * plan->nf3, hipMemcpyDeviceToHost);
        for (int i = 0; i < 10; i++)
        {
                printf("%.3g ", fw[i].real());
            printf("\n");
        }
#endif
        // keep the N1*N2*num_w. ignore the outputs that are out of range

        // 3. dft on w (or 1 dimensional nufft type3)
#ifdef TIME
        hipEventRecord(start);
#endif
        if(!plan->mem_limit)curadft_invoker(plan, gridder_plan->pixelsize_x, gridder_plan->pixelsize_y);
#ifdef TIME
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        totaltime += milliseconds;
        printf("[time  ] dft w time:\t\t %.3g s\n", milliseconds / 1000);
#endif
#ifdef DEBUG
        printf("part of dft result printing:...\n");
        //CPX *fw = (CPX *)malloc(sizeof(CPX)*plan->nf1*plan->nf2*plan->nf3);
        hipMemcpy(fw, plan->fw, sizeof(CUCPX) * plan->nf1 * plan->nf2 * plan->nf3, hipMemcpyDeviceToHost);
        for (int i = 0; i < 10; i++)
        {
            
                printf("%.3g ", fw[i ].real());
            printf("\n");
        }
#endif
        if(plan->mem_limit){
                int nhive[3];
                nhive[0] = (plan->nf1-1)/plan->hivesize[0] + 1;
                nhive[1] = (plan->nf2-1)/plan->hivesize[1] + 1;
                nhive[2] = (plan->nf3-1)/plan->hivesize[2] + 1;
                unsigned long int histo_count_size = nhive[0]*plan->hivesize[0]; // padding
                histo_count_size *= nhive[1]*plan->hivesize[1];
                histo_count_size *= nhive[2]*plan->hivesize[2];
                histo_count_size ++;
                int i;
                int up_shift, c_shift, down_shift;
                for(i=0; i<(plan->mem_limit-1)/plan->nf3; i++){
                        show_mem_usage();
                        checkCudaErrors(hipFree(plan->fw));
                        checkCudaErrors(hipMalloc((void **)&plan->histo_count,sizeof(int)*(histo_count_size)));
                        checkCudaErrors(hipMemset(plan->histo_count,0,sizeof(int)*(histo_count_size)));
                        part_bin_mapping(plan, plan->d_u_out, plan->d_v_out, plan->d_w_out, plan->d_c_out, histo_count_size, i+1, plan->initial);
                        checkCudaErrors(hipFree(plan->histo_count));
                        checkCudaErrors(hipFree(plan->sortidx_bin));
                        checkCudaErrors(hipMalloc((void **)&plan->fw, sizeof(CUCPX) * plan->nf1 * plan->nf2 * plan->nf3));
                        checkCudaErrors(hipMemset(plan->fw, 0, plan->nf3 * plan->nf1 * plan->nf2 * sizeof(CUCPX)));
                        if(i%2){
                                c_shift = nhive[0]*nhive[1]*nhive[2]+1;
                                down_shift = 0;
                        }
                        else{
                                c_shift = 0;
                                down_shift = nhive[0]*nhive[1]*nhive[2]+1;
                        }
                        up_shift = nhive[0]*nhive[1]*nhive[2]*2+2;
                        int remain_batch = curaff_partial_conv(plan, i*plan->nf3, up_shift, c_shift, down_shift);

                        show_mem_usage();
                        // cufft plan setting
                        cufft_plan_setting(plan);
                        cura_cufft(plan);
                
                        hipfftDestroy(plan->fftplan);
                        if(remain_batch!=0) hipfftDestroy(plan->fftplan_l);
                        
                        curadft_partial_invoker(plan, gridder_plan->pixelsize_x, gridder_plan->pixelsize_y, i*plan->nf3);

                }
                // last cube
                checkCudaErrors(hipMemcpy(plan->hive_count+nhive[0]*nhive[1]*nhive[2]*2+2,gridder_plan->temp_station,sizeof(int)*(nhive[0]*nhive[1]+1),hipMemcpyHostToDevice));
                free(gridder_plan->temp_station);
                
                checkCudaErrors(hipMemset(plan->fw, 0, plan->nf3 * plan->nf1 * plan->nf2 * sizeof(CUCPX)));

                int nf3 = plan->nf3;
                plan->nf3 = plan->mem_limit - i * nf3;
                cufft_plan_setting(plan);

                if(i%2){
                        c_shift = nhive[0]*nhive[1]*nhive[2]+1;
                        up_shift = nhive[0]*nhive[1]*nhive[2] - nhive[0]*nhive[1];
                }
                else{
                        c_shift = 0;
                        up_shift = nhive[0]*nhive[1]*nhive[2]*2+1 - nhive[0]*nhive[1];
                }
                down_shift = nhive[0]*nhive[1]*nhive[2]*2+2;
                int remain_batch = curaff_partial_conv(plan, i*nf3, up_shift, c_shift, down_shift);
                
                cura_cufft(plan);
                if(remain_batch!=0){
                        hipfftDestroy(plan->fftplan_l);
                }
                curadft_partial_invoker(plan, gridder_plan->pixelsize_x, gridder_plan->pixelsize_y, i*nf3);
                checkCudaErrors(hipFree(plan->fw));
                plan->fw = plan->fw_temp;
        }
        // 4. deconvolution (correction)
        // error detected, 1. w term deconv
        // 1. 2D deconv towards u and v
#ifdef TIME
        hipEventRecord(start);
#endif
        plan->dim = 2;
        ier = curafft_deconv(plan);
#ifdef TIME
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        totaltime += milliseconds;
        printf("[time  ] 2d deconv time:\t\t %.3g s\n", milliseconds / 1000);
#endif
#ifdef DEBUG
        printf("deconv result printing stage 1:...\n");
        CPX *fk = (CPX *)malloc(sizeof(CPX) * plan->ms * plan->mt);
        hipMemcpy(fk, plan->fk, sizeof(CUCPX) * plan->ms * plan->mt, hipMemcpyDeviceToHost);
        for (int i = 0; i < 10; i++)
        {
            
                printf("%.5lf ", fk[i].real());
            printf("\n");
        }
#endif
        // 2. w term deconv on fk
#ifdef TIME
        hipEventRecord(start);
#endif
        ier = curadft_w_deconv(plan, gridder_plan->pixelsize_x, gridder_plan->pixelsize_y);
#ifdef TIME
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        totaltime += milliseconds;
        printf("[time  ] w term deconv time:\t\t %.3g s\n", milliseconds / 1000);
#endif
#ifdef DEBUG
        printf("deconv result printing stage 2:...\n");
        //CPX *fk = (CPX *)malloc(sizeof(CPX)*plan->ms*plan->mt);
        hipMemcpy(fk, plan->fk, sizeof(CUCPX) * plan->ms * plan->mt, hipMemcpyDeviceToHost);
        for (int i = 0; i < 10; i++)
        {
            
                printf("%.5lf ", fk[i].real());
            printf("\n");
        }
#endif
        // 5. ending work - scaling
        // /n_lm, fourier related rescale
#ifdef TIME
        hipEventRecord(start);
#endif
        cura_rscaling(plan, gridder_plan);
#ifdef TIME
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        totaltime += milliseconds;
        printf("[time  ] end work time:\t\t %.3g s\n", milliseconds / 1000);
        printf("[time  ] Total time:\t\t %.3g s\n", totaltime/1000);
#endif    
    }
    else if (plan->execute_flow == 2)
    {
        /// curafft_partial_conv workflow for insufficient memory

        // offset array with size of
        for (int i = 0; i < gridder_plan->num_w; i += plan->batchsize)
        {
            //memory allocation of fw may cause error, if size is too large, decrease the batchsize.
            checkCudaErrors(hipMemset(plan->fw, 0, plan->batchsize * plan->nf1 * plan->nf2 * sizeof(CUCPX)));
            // 1. convlution
            curafft_conv(plan);
        }
    }
    return ier;
}

int exec_dirty2vis(CURAFFT_PLAN *plan, ragridder_plan *gridder_plan){
    int ier=0;
    

    cura_prestage(plan,gridder_plan);

    // 1. scaling (/n) 
    cura_rscaling(plan, gridder_plan);

    // 2. deconvolution
    // 2.1 w term
    ier = curadft_w_deconv(plan, gridder_plan->pixelsize_x, gridder_plan->pixelsize_y); // fk * e-

#ifdef DEBUG
    printf("deconv result printing stage 2:...\n");
        CPX *fk = (CPX *)malloc(sizeof(CPX)*plan->ms*plan->mt);
        hipMemcpy(fk, plan->fk, sizeof(CUCPX) * plan->ms * plan->mt, hipMemcpyDeviceToHost);
        for(int j=0; j<plan->mt; j++){
                for (int i = 0; i < plan->ms; i++)
        {
                printf("%.5g ", fk[i+plan->ms*j].real());   
        }
        printf("\n");
        }
        
        free(fk);
#endif  
    // 2.2 2D deconv towards u and v
    plan->dim = 2;
    ier = curafft_deconv(plan); // need to revise, back to FFTW mode?
    plan->dim = 3;
#ifdef DEBUG
        printf("deconv result printing\n");
        CPX *fw = (CPX *)malloc(sizeof(CPX)*plan->nf1*plan->nf2*plan->nf3);
        hipMemcpy(fw, plan->fw, sizeof(CUCPX) * plan->nf1 * plan->nf2 * plan->nf3, hipMemcpyDeviceToHost);
        for (int j = 0; j<1; j++){
                printf("plane %d begin..............................................................\n",j);
                for (int i = 0; i <  plan->nf1 * plan->nf2; i++)
                {
                        printf("%.6g ", fw[i+j*plan->nf1*plan->nf2].real());
                        if(i%plan->nf1==0)printf("\n");
                        
                }
                printf("\n");
        }
        free(fw);
#endif
    // 3. idft
    curadft_invoker(plan, gridder_plan->pixelsize_x, gridder_plan->pixelsize_y);


#ifdef DEBUG
     printf("idft result printing\n");
        CPX *fw = (CPX *)malloc(sizeof(CPX)*plan->nf1*plan->nf2*plan->nf3);
        hipMemcpy(fw, plan->fw, sizeof(CUCPX) * plan->nf1 * plan->nf2 * plan->nf3, hipMemcpyDeviceToHost);
        for (int j = 0; j<plan->nf3; j++){
                printf("plane %d begin..............................................................\n",j);
                for (int i = plan->nf1 * plan->nf2-plan->nf2-2; i <  plan->nf1 * plan->nf2-plan->nf2-1; i++)
                {
                        if(i%plan->nf1==0)printf("\n");
                        printf("%.10g ", fw[i+j*plan->nf1*plan->nf2].real());
                        
                        
                }
                printf("\n");
        }
#endif
        //free(fw);

    // 4. fft
        cura_cufft(plan);
#ifdef DEBUG
        printf("fft result printing\n");
        //CPX *fw = (CPX *)malloc(sizeof(CPX)*plan->nf1*plan->nf2*plan->nf3);
        hipMemcpy(fw, plan->fw, sizeof(CUCPX) * plan->nf1 * plan->nf2 * plan->nf3, hipMemcpyDeviceToHost);
        printf("%d,%d,%d\n",plan->nf1,plan->nf2,plan->nf3);
        int count=0;
        for (int j = 0; j<plan->nf3; j++){
                printf("plane %d begin..............................................................\n",j);
                for (int i = 0; i < plan->nf1*plan->nf2; i++)
                {
                        if(fw[i+j*plan->nf1*plan->nf2].real()==0)
                        count++;
                        
                }
                printf("\n");
        }
        printf("count: %d\n",count);
        
        CPX *c = (CPX *)malloc(sizeof(CPX) * plan->M);
        //CPX *fw = (CPX *)malloc(sizeof(CPX)*plan->nf1*plan->nf2*plan->nf3);
        hipMemcpy(c, plan->d_c, sizeof(CUCPX) * plan->M, hipMemcpyDeviceToHost);

#endif
        // * e-izw_j
    // 5. interpolation
    curafft_interp(plan);
#ifdef DEBUG
        printf("interp result printing (first w plane)...\n");
        hipMemcpy(c, plan->d_c, sizeof(CUCPX) * plan->M, hipMemcpyDeviceToHost);
        PCS temp = 0;
        for (int i = 0; i < 100; i++)
        {
                temp += c[i].real();
                printf("%.6g ", c[i].real());
        }
        free(c);
#endif
    // 6. final work (/wgt, *e)
    cura_fw(plan,gridder_plan); // 
    return ier;
}