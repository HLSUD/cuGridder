#include "hip/hip_runtime.h"
/*
  Utility functions
  1. prefix_scan
  2. get_max_min
  3. rescale
  4. shift_and_scale
  5. matrix transpose
*/

#include "utils.h"
#include "common_utils.h"
#include <cstdlib>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <stdio.h>
//#include <thrust/extrema.h>
#include <thrust/device_ptr.h>
#include <thrust/scan.h>
#include <hipcub/hipcub.hpp>
#include "datatype.h"
#include "curafft_plan.h"
#include "conv.h"

void prefix_scan(PCS *d_arr, PCS *d_res, int n, int flag)
{
  /*
    n - number of elements
    flag - 1 inclusive, 0 exclusive
    thrust::inclusive_scan(d_arr, d_arr + n, d_res);
  */
  thrust::device_ptr<PCS> d_ptr(d_arr); // not convert
  thrust::device_ptr<PCS> d_result(d_res);

  if (flag)
    thrust::inclusive_scan(d_ptr, d_ptr + n, d_result);
  else
    thrust::exclusive_scan(d_ptr, d_ptr + n, d_result);
}

void get_max_min(PCS &max, PCS &min, PCS *d_array, int n)
{
  /*
    Get the maximum and minimum of array by thrust
    d_array - array on device
    n - length of array
  */
  thrust::device_ptr<PCS> d_ptr = thrust::device_pointer_cast(d_array);
  max = *(thrust::max_element(d_ptr, d_ptr + n));

  min = *(thrust::min_element(d_ptr, d_ptr + n));
}

// __global__ part_histogram_3d(int N_v, int N_b, int scounter){
//   /*
//   N_v - number of points that need to map
//   N_b - number of bins
//   scounter - record start point
//   */
//   int idx;
//   for (idx = threadIdx.x + blockIdx.x * blockDim.x; idx < N_v; idx+=blockDim.x*gridDim.x){
//     // get bin index
//     int bindex = 0;
    
//     // shared memory allocation and initate to 0

//     // 
//   }
// }

__global__ void part_histogram_3d_sparse(PCS *x, PCS *y, PCS *z, int *sortidx_bin, int *histo_count,
    int N_v, int nf1, int nf2, int nf3, int plane, int pirange){
  /*
  do not use privitization due to sparsity. 
  histogram one plane by one plane (limitation of memory) ++++ sorted
  */
  int idx;
  int bin_x, bin_y, bin_z;
  for (idx = threadIdx.x + blockIdx.x * blockDim.x; idx < N_v; idx+=blockDim.x*gridDim.x){
    // get bin index
    bin_x = floor(SHIFT_RESCALE(x[idx], nf1, pirange));
		bin_y = floor(SHIFT_RESCALE(y[idx], nf2, pirange));
		bin_z = floor(SHIFT_RESCALE(z[idx], nf3, pirange));
    /// 2d or somehow 3d partical
    if(bin_z==plane){
      int bindex = bin_y * nf1 + bin_x;
      int old = atomicAdd(&histo_count[bindex],1);
      sortidx_bin[idx] = old;
    }
  }
}

void part_histogram_3d_sparse_invoker(PCS *x, PCS *y, PCS *z, int *sortidx_bin, int *histo_count, int N_v, int nf1, int nf2, int nf3, int plane, int pirange){
  int blocksize = 256;
  part_histogram_3d_sparse<<<(N_v-1)/blocksize+1,blocksize>>>(x,y,z,sortidx_bin,histo_count,N_v,nf1,nf2,nf3,plane,pirange);
  checkCudaErrors(hipDeviceSynchronize());
}

__global__ void histogram_3d_sparse(PCS *x, PCS *y, PCS *z, int *sortidx_bin, int *histo_count,
    int N_v, int nf1, int nf2, int nf3, int hivesize_x, int hivesize_y, int hivesize_z, int nhive_x, int nhive_y, int nhive_z, int pirange){
  /*
  do not use privitization due to sparsity. 
  */
  // found the reason
  
  int idx;
  int bin_x, bin_y, bin_z;
  unsigned long int histo_idx;
  for (idx = threadIdx.x + blockIdx.x * blockDim.x; idx < N_v; idx+=blockDim.x*gridDim.x){
    // get bin index
    bin_x = floor(SHIFT_RESCALE(x[idx], nf1, pirange));
		bin_y = floor(SHIFT_RESCALE(y[idx], nf2, pirange));
		bin_z = floor(SHIFT_RESCALE(z[idx], nf3, pirange));
    int hive_x = bin_x / hivesize_x;
    int hive_y = bin_y / hivesize_y;
    int hive_z = bin_z / hivesize_z;
    histo_idx = hive_x + hive_y * nhive_x + hive_z * nhive_x * nhive_y;
    histo_idx *= hivesize_x * hivesize_y * hivesize_z;
    histo_idx += bin_x % hivesize_x + (bin_y % hivesize_y) * hivesize_x + (bin_z % hivesize_z) * hivesize_x * hivesize_y;
    // printf("%d,%d,%d\n",hive_x,hive_y,hive_z);
    int old = atomicAdd(&histo_count[histo_idx],1);
    sortidx_bin[idx] = old;
    
  }
}

__global__ void final_hive_plane_histo(PCS *x, PCS *y, PCS *z, int *sortidx_bin, int *histo_count,
    int N_v, int nf1, int nf2, int nf3, int hivesize_x, int hivesize_y, int hivesize_z, 
    int nhive_x, int nhive_y, int nhive_z, int plane, int pirange){
  int idx;
  int bin_x, bin_y, bin_z;
  unsigned long int histo_idx;
  for (idx = threadIdx.x + blockIdx.x * blockDim.x; idx < N_v; idx+=blockDim.x*gridDim.x){
    // get bin index
    bin_x = floor(SHIFT_RESCALE(x[idx], nf1, pirange));
		bin_y = floor(SHIFT_RESCALE(y[idx], nf2, pirange));
		bin_z = floor(SHIFT_RESCALE(z[idx], nf3, pirange));
    int hive_x = bin_x / hivesize_x;
    int hive_y = bin_y / hivesize_y;
    // int hive_z = bin_z / hivesize_z;
    // cube_id
    if(bin_z >= plane){ // constant nf3->nhive
      histo_idx = hive_x + hive_y * nhive_x;
      histo_idx *= hivesize_x * hivesize_y * hivesize_z;
      histo_idx += bin_x % hivesize_x + (bin_y % hivesize_y) * hivesize_x + (bin_z % hivesize_z) * hivesize_x * hivesize_y;
      // printf("%d,%d,%d\n",hive_x,hive_y,hive_z);
      int old = atomicAdd(&histo_count[histo_idx],1);
      sortidx_bin[idx] = old;
    }
  }
}
__global__ void final_hive_mapping_gather(PCS *x, PCS *y, PCS *z, CUCPX *c, PCS *x_out, PCS *y_out, PCS *z_out, CUCPX *c_out,
    int *sortidx_bin, int *histo_count, int N_v, int nf1, int nf2, int nf3, int hivesize_x, int hivesize_y, int hivesize_z, 
    int nhive_x, int nhive_y, int nhive_z, int plane, int total, int pirange){
  int idx;
  PCS x1, y1, z1;
  unsigned long int histo_idx;
  for (idx = threadIdx.x + blockIdx.x * blockDim.x; idx < N_v; idx+=blockDim.x*gridDim.x){
    // get bin index
    x1 = SHIFT_RESCALE(x[idx], nf1, pirange);
		y1 = SHIFT_RESCALE(y[idx], nf2, pirange);
		z1 = SHIFT_RESCALE(z[idx], nf3, pirange);
    int bin_x = floor(x1);
    int bin_y = floor(y1);
    int bin_z = floor(z1);
    int hive_x = bin_x / hivesize_x;
    int hive_y = bin_y / hivesize_y;
    // int hive_z = bin_z / hivesize_z;
    // if(abs(x[idx]-0.152601)<0.0001)printf("---%d,%d,%d\n",hive_x,hive_y,hive_z);
    if(bin_z>=plane){
      histo_idx = hive_x + hive_y * nhive_x;
      histo_idx *= hivesize_x * hivesize_y * hivesize_z;
      histo_idx += bin_x % hivesize_x + (bin_y % hivesize_y) * hivesize_x + (bin_z % hivesize_z) * hivesize_x * hivesize_y;

      int loc = N_v - (total - sortidx_bin[idx]-histo_count[histo_idx]);
      // if(abs(x[idx]-0.152601)<0.0001)printf("-------loc %d\n",loc);
      x_out[loc] = x1;
      y_out[loc] = y1;
      z_out[loc] = z1;
      c_out[loc] = c[idx];
    }
  }
}

void final_hive_plane_bin_mapping(PCS *x, PCS *y, PCS *z, CUCPX *c, PCS *x_out, PCS *y_out, PCS *z_out, CUCPX *c_out,
    int *sortidx_bin, int *histo_count, int *hive_count, int N_v, int nf1, int nf2, int nf3, int *hivesize, int* nhive, int pirange){
  int plane = nf3 - 8;
  int blocksize = 256;
  final_hive_plane_histo<<<(N_v-1)/blocksize+1,blocksize>>>(x,y,z,sortidx_bin,histo_count,N_v,nf1,nf2,nf3,hivesize[0], hivesize[1], hivesize[2], nhive[0], nhive[1], nhive[2], plane, pirange);
  checkCudaErrors(hipDeviceSynchronize());
  prefix_scan(histo_count,histo_count,nhive[0]*nhive[1]*hivesize[0]*hivesize[1]*hivesize[2]+1,0);
  int total;
  checkCudaErrors(hipMemcpy(&total,histo_count+nhive[0]*nhive[1]*hivesize[0]*hivesize[1]*hivesize[2],sizeof(int),hipMemcpyDeviceToHost));
  // count
  counting_hive_invoker(hive_count,histo_count,nhive[0]*nhive[1]+1,hivesize[0]*hivesize[1]*hivesize[2],N_v-total);
  final_hive_mapping_gather<<<(N_v-1)/blocksize+1,blocksize>>>(x,y,z,c,x_out,y_out,z_out,c_out,sortidx_bin,histo_count,N_v,nf1,nf2,nf3,hivesize[0],hivesize[1],hivesize[2],nhive[0],nhive[1],nhive[2],plane,total,pirange);
  checkCudaErrors(hipDeviceSynchronize());
}


__global__ void part_histogram_3d_cube(PCS *x, PCS *y, PCS *z, int *sortidx_bin, int *histo_count,
    int N_v, int nf1, int nf2, int nf3, int hivesize_x, int hivesize_y, int hivesize_z, 
    int nhive_x, int nhive_y, int nhive_z, int cube_id, int cube_z, int pirange){
  /*
  do not use privitization due to sparsity. 
  */
  // found the reason
  
  int idx;
  int bin_x, bin_y, bin_z;
  unsigned long int histo_idx;
  for (idx = threadIdx.x + blockIdx.x * blockDim.x; idx < N_v; idx+=blockDim.x*gridDim.x){
    // get bin index
    bin_x = floor(SHIFT_RESCALE(x[idx], nf1, pirange));
		bin_y = floor(SHIFT_RESCALE(y[idx], nf2, pirange));
		bin_z = floor(SHIFT_RESCALE(z[idx], nf3, pirange));
    int hive_x = bin_x / hivesize_x;
    int hive_y = bin_y / hivesize_y;
    int hive_z = bin_z / hivesize_z;
    // cube_id
    if(bin_z/cube_z==cube_id){ // constant nf3->nhive
      histo_idx = hive_x + hive_y * nhive_x + (hive_z-cube_id*cube_z/8) * nhive_x * nhive_y;
      histo_idx *= hivesize_x * hivesize_y * hivesize_z;
      histo_idx += bin_x % hivesize_x + (bin_y % hivesize_y) * hivesize_x + (bin_z % hivesize_z) * hivesize_x * hivesize_y;
      // printf("%d,%d,%d\n",hive_x,hive_y,hive_z);
      int old = atomicAdd(&histo_count[histo_idx],1);
      sortidx_bin[idx] = old;
    }
  }
}

void histogram_3d_cube_invoker(PCS *x, PCS *y, PCS *z, int *sortidx_bin, int *histo_count,
    int N_v, int nf1, int nf2, int nf3, int *hivesize, int* nhive, int cube_id, int cube_z, int pirange){
  int blocksize = 512;
  part_histogram_3d_cube<<<(N_v-1)/blocksize+1,blocksize>>>(x,y,z,sortidx_bin,histo_count,N_v,nf1,nf2,nf3,hivesize[0], hivesize[1], hivesize[2], nhive[0], nhive[1], nhive[2], cube_id, cube_z, pirange);
  checkCudaErrors(hipDeviceSynchronize());
}

void histogram_3d_sparse_invoker(PCS *x, PCS *y, PCS *z, int *sortidx_bin, int *histo_count, int N_v, int nf1, int nf2, int nf3, int *hivesize, int* nhive, int pirange){
  int blocksize = 512;
  histogram_3d_sparse<<<(N_v-1)/blocksize+1,blocksize>>>(x,y,z,sortidx_bin,histo_count,N_v,nf1,nf2,nf3,hivesize[0], hivesize[1], hivesize[2], nhive[0], nhive[1], nhive[2], pirange);
  checkCudaErrors(hipDeviceSynchronize());
}

__global__ void part_mapping_based_gather_3d(PCS *x, PCS *y, PCS *z, CUCPX *c, PCS *x_out, PCS *y_out, PCS *z_out, CUCPX *c_out,
    int *sortidx_bin, int *histo_count, int2 *se_loc, int N_v, int nf1, int nf2, int nf3, int plane, int init_scan_value, int pirange){
  int idx;
  int temp1, temp2, temp3;
  for (idx = threadIdx.x + blockIdx.x * blockDim.x; idx < N_v; idx+=blockDim.x*gridDim.x){
    // get bin index
    temp1 = floor(SHIFT_RESCALE(x[idx], nf1, pirange));
		temp2 = floor(SHIFT_RESCALE(y[idx], nf2, pirange));
		temp3 = floor(SHIFT_RESCALE(z[idx], nf3, pirange));
    if(temp3==plane){
      int bindex = temp2 * nf1 + temp1;
      int start_loc = histo_count[bindex]+init_scan_value;
      int loc = sortidx_bin[idx]+start_loc;
      x_out[loc] = x[idx];
      y_out[loc] = y[idx];
      z_out[loc] = z[idx];
      c_out[loc] = c[idx];
      se_loc[loc].x = histo_count[bindex]+init_scan_value;
      se_loc[loc].y = histo_count[bindex+1]+init_scan_value;
    }
  }
}

void part_mapping_based_gather_3d_invoker(PCS *x, PCS *y, PCS *z, CUCPX *c, PCS *x_out, PCS *y_out, PCS *z_out, CUCPX *c_out,
    int *sortidx_bin, int *histo_count, int2 *se_loc, int N_v, int nf1, int nf2, int nf3, int plane, int init_scan_value, int pirange){
  int blocksize = 256;
  part_mapping_based_gather_3d<<<(N_v-1)/blocksize+1,blocksize>>>(x,y,z,c,x_out,y_out,z_out,c_out,sortidx_bin,histo_count,se_loc,N_v,nf1,nf2,nf3,plane,init_scan_value,pirange);
  checkCudaErrors(hipDeviceSynchronize());
}

__global__ void part_mapping_based_gather_3d(PCS *x, PCS *y, PCS *z, CUCPX *c, PCS *x_out, PCS *y_out, PCS *z_out, CUCPX *c_out,
    int *sortidx_bin, int *histo_count, int N_v, int nf1, int nf2, int nf3, int hivesize_x, int hivesize_y, int hivesize_z, 
    int nhive_x, int nhive_y, int nhive_z, int cube_id, int cube_z, int init_scan_value, int pirange){
  // replace issue
  int idx;
  PCS x1, y1, z1;
  unsigned long int histo_idx;
  for (idx = threadIdx.x + blockIdx.x * blockDim.x; idx < N_v; idx+=blockDim.x*gridDim.x){
    // get bin index
    x1 = SHIFT_RESCALE(x[idx], nf1, pirange);
		y1 = SHIFT_RESCALE(y[idx], nf2, pirange);
		z1 = SHIFT_RESCALE(z[idx], nf3, pirange);
    int bin_x = floor(x1);
    int bin_y = floor(y1);
    int bin_z = floor(z1);
    int hive_x = bin_x / hivesize_x;
    int hive_y = bin_y / hivesize_y;
    int hive_z = bin_z / hivesize_z;
    // if(abs(x[idx]-0.152601)<0.0001)printf("---%d,%d,%d\n",hive_x,hive_y,hive_z);
    if(bin_z/cube_z==cube_id){
      histo_idx = hive_x + hive_y * nhive_x + (hive_z-cube_id*cube_z/8) * nhive_x * nhive_y;
      histo_idx *= hivesize_x * hivesize_y * hivesize_z;
      histo_idx += bin_x % hivesize_x + (bin_y % hivesize_y) * hivesize_x + (bin_z % hivesize_z) * hivesize_x * hivesize_y;

      int loc = sortidx_bin[idx]+histo_count[histo_idx]+init_scan_value;
      // if(abs(x[idx]-0.152601)<0.0001)printf("-------loc %d\n",loc);
      x_out[loc] = x1;
      y_out[loc] = y1;
      z_out[loc] = z1;
      c_out[loc] = c[idx];
    }
  }
}

void part_mapping_based_gather_3d_invoker(PCS *x, PCS *y, PCS *z, CUCPX *c, PCS *x_out, PCS *y_out, PCS *z_out, CUCPX *c_out,
    int *sortidx_bin, int *histo_count, int N_v, int nf1, int nf2, int nf3, int *hivesize, int *nhive, int cube_id, int cube_z, 
    int init_scan_value, int pirange){
  int blocksize = 256;
  part_mapping_based_gather_3d<<<(N_v-1)/blocksize+1,blocksize>>>(x,y,z,c,x_out,y_out,z_out,c_out,sortidx_bin,histo_count,N_v,nf1,nf2,nf3,hivesize[0],hivesize[1],hivesize[2],nhive[0],nhive[1],nhive[2],cube_id,cube_z,init_scan_value,pirange);
  checkCudaErrors(hipDeviceSynchronize());
}

__global__ void mapping_based_gather_3d(PCS *x, PCS *y, PCS *z, CUCPX *c, PCS *x_out, PCS *y_out, PCS *z_out, CUCPX *c_out,
    int *sortidx_bin, int *histo_count, int N_v, int nf1, int nf2, int nf3, int hivesize_x, int hivesize_y, int hivesize_z, int nhive_x, int nhive_y, int nhive_z, int pirange){
  int idx;
  int bin_x, bin_y, bin_z;
  unsigned long int histo_idx;
  for (idx = threadIdx.x + blockIdx.x * blockDim.x; idx < N_v; idx+=blockDim.x*gridDim.x){
    // get bin index
    bin_x = floor(SHIFT_RESCALE(x[idx], nf1, pirange));
		bin_y = floor(SHIFT_RESCALE(y[idx], nf2, pirange));
		bin_z = floor(SHIFT_RESCALE(z[idx], nf3, pirange));

    int hive_x = bin_x / hivesize_x;
    int hive_y = bin_y / hivesize_y;
    int hive_z = bin_z / hivesize_z;
    // if(abs(x[idx]-0.152601)<0.0001)printf("---%d,%d,%d\n",hive_x,hive_y,hive_z);
    histo_idx = hive_x + hive_y * nhive_x + hive_z * nhive_x * nhive_y;
    histo_idx *= hivesize_x * hivesize_y * hivesize_z;
    histo_idx += bin_x % hivesize_x + (bin_y % hivesize_y) * hivesize_x + (bin_z % hivesize_z) * hivesize_x * hivesize_y;

    int loc = sortidx_bin[idx]+histo_count[histo_idx];
    // if(abs(x[idx]-0.152601)<0.0001)printf("-------loc %d\n",loc);
    x_out[loc] = x[idx];
    y_out[loc] = y[idx];
    z_out[loc] = z[idx];
    c_out[loc] = c[idx];
  }
}

__global__ void mapping_based_gather_3d_replace(PCS *x, PCS *y, PCS *z, CUCPX *c, PCS *x_out, PCS *y_out, PCS *z_out, CUCPX *c_out,
    int *sortidx_bin, int *histo_count, int N_v, int nf1, int nf2, int nf3, int hivesize_x, int hivesize_y, int hivesize_z, int nhive_x, int nhive_y, int nhive_z, int pirange){
  int idx;
  PCS x1, y1, z1;
  unsigned long int histo_idx;
  for (idx = threadIdx.x + blockIdx.x * blockDim.x; idx < N_v; idx+=blockDim.x*gridDim.x){
    // get bin index
    x1 = SHIFT_RESCALE(x[idx], nf1, pirange);
		y1 = SHIFT_RESCALE(y[idx], nf2, pirange);
		z1 = SHIFT_RESCALE(z[idx], nf3, pirange);
    int bin_x = floor(x1);
    int bin_y = floor(y1);
    int bin_z = floor(z1);
    int hive_x = bin_x / hivesize_x;
    int hive_y = bin_y / hivesize_y;
    int hive_z = bin_z / hivesize_z;
    // if(abs(x[idx]-0.152601)<0.0001)printf("---%d,%d,%d\n",hive_x,hive_y,hive_z);
    histo_idx = hive_x + hive_y * nhive_x + hive_z * nhive_x * nhive_y;
    histo_idx *= hivesize_x * hivesize_y * hivesize_z;
    histo_idx += bin_x % hivesize_x + (bin_y % hivesize_y) * hivesize_x + (bin_z % hivesize_z) * hivesize_x * hivesize_y;

    int loc = sortidx_bin[idx]+histo_count[histo_idx];
    // if(abs(x[idx]-0.152601)<0.0001)printf("-------loc %d\n",loc);
    x_out[loc] = x1;
    y_out[loc] = y1;
    z_out[loc] = z1;
    c_out[loc] = c[idx];
  }
}

void mapping_based_gather_3d_invoker(PCS *x, PCS *y, PCS *z, CUCPX *c, PCS *x_out, PCS *y_out, PCS *z_out, CUCPX *c_out,
    int *sortidx_bin, int *histo_count, int N_v, int nf1, int nf2, int nf3, int *hivesize, int* nhive, int method, int pirange){
  int blocksize = 256;
  if(method==2)
  mapping_based_gather_3d<<<(N_v-1)/blocksize+1,blocksize>>>(x,y,z,c,x_out,y_out,z_out,c_out,sortidx_bin,histo_count,N_v,nf1,nf2,nf3,hivesize[0], hivesize[1], hivesize[2], nhive[0], nhive[1], nhive[2], pirange);
  if(method==3||method==4)mapping_based_gather_3d_replace<<<(N_v-1)/blocksize+1,blocksize>>>(x,y,z,c,x_out,y_out,z_out,c_out,sortidx_bin,histo_count,N_v,nf1,nf2,nf3,hivesize[0], hivesize[1], hivesize[2], nhive[0], nhive[1], nhive[2], pirange);
  checkCudaErrors(hipDeviceSynchronize());
}

// real and complex array scaling
__global__ void rescaling_real(PCS *x, PCS scale_ratio, int N)
{
  int idx;
  for (idx = blockIdx.x * blockDim.x + threadIdx.x; idx < N; idx += gridDim.x * blockDim.x)
  {
    x[idx] *= scale_ratio;
  }
}

__global__ void rescaling_complex(CUCPX *x, PCS scale_ratio, int N)
{
  int idx;
  for (idx = blockIdx.x * blockDim.x + threadIdx.x; idx < N; idx += gridDim.x * blockDim.x)
  {
    x[idx].x *= scale_ratio;
    x[idx].y *= scale_ratio;
  }
}

void rescaling_real_invoker(PCS *d_x, PCS scale_ratio, int N)
{
  int blocksize = 512;
  rescaling_real<<<(N - 1) / blocksize + 1, blocksize>>>(d_x, scale_ratio, N);
  CHECK(hipDeviceSynchronize());
}

void rescaling_complex_invoker(CUCPX *d_x, PCS scale_ratio, int N)
{
  int blocksize = 512;
  rescaling_complex<<<(N - 1) / blocksize + 1, blocksize>>>(d_x, scale_ratio, N);
  CHECK(hipDeviceSynchronize());
}

__global__ void shift_and_scale(PCS i_center, PCS o_center, PCS gamma, PCS *d_u, PCS *d_x, int M, int N)
{
  int idx;
  for (idx = blockIdx.x * blockDim.x + threadIdx.x; idx < M; idx += gridDim.x * blockDim.x)
  {
    d_u[idx] = (d_u[idx] - i_center) / gamma;
  }
  for (idx = blockIdx.x * blockDim.x + threadIdx.x; idx < N; idx += gridDim.x * blockDim.x)
  {
    d_x[idx] = (d_x[idx] - o_center) * gamma;
  }
}

void shift_and_scale_invoker(PCS i_center, PCS o_center, PCS gamma, PCS *d_u, PCS *d_x, int M, int N)
{
  // Specified for nu to nu fourier transform
  int blocksize = 512;
  shift_and_scale<<<(max(M, N) - 1) / blocksize + 1, blocksize>>>(i_center, o_center, gamma, d_u, d_x, M, N);
  CHECK(hipDeviceSynchronize());
}

__global__ void transpose(PCS *odata, PCS *idata, int width, int height)
{
  //* Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
  // refer https://github.com/JonathanWatkins/CUDA/blob/master/NvidiaCourse/Exercises/transpose/transpose.cu
  __shared__ PCS block[BLOCKSIZE][BLOCKSIZE];

  // read the matrix tile into shared memory
  // load one element per thread from device memory (idata) and store it
  // in transposed order in block[][]
  unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x; //height
  unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y; //width
  if ((yIndex < width) && (xIndex < height))
  {
    unsigned int index_in = xIndex * width + yIndex;
    block[threadIdx.x][threadIdx.y] = idata[index_in];
  }

  // synchronise to ensure all writes to block[][] have completed
  __syncthreads();

  // write the transposed matrix tile to global memory (odata) in linear order
  xIndex = blockIdx.y * blockDim.x + threadIdx.x;
  yIndex = blockIdx.x * blockDim.y + threadIdx.y;
  if ((yIndex < height) && (xIndex < width))
  {
    unsigned int index_out = xIndex * height + yIndex;
    odata[index_out] = block[threadIdx.y][threadIdx.x];
  }
  // __syncthreads();
}

int matrix_transpose_invoker(PCS *d_arr, int width, int height)
{
  int ier = 0;
  int blocksize = BLOCKSIZE;
  dim3 block(blocksize, blocksize);
  dim3 grid((height - 1) / blocksize + 1, (width - 1) / blocksize + 1);
  PCS *temp_o;
  checkCudaErrors(hipMalloc((void **)&temp_o, sizeof(PCS) * width * height));
  transpose<<<grid, block>>>(temp_o, d_arr, width, height);
  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipMemcpy(d_arr, temp_o, sizeof(PCS) * width * height, hipMemcpyDeviceToDevice));
  checkCudaErrors(hipFree(temp_o));
  return ier;
}

__global__ void matrix_elementwise_multiply(CUCPX *a, PCS *b, int N)
{
  int idx;
  for (idx = threadIdx.x + blockIdx.x * blockDim.x; idx < N; idx += gridDim.x * blockDim.x)
  {
    a[idx].x = a[idx].x * b[idx];
    a[idx].y = a[idx].y * b[idx];
  }
}

int matrix_elementwise_multiply_invoker(CUCPX *a, PCS *b, int N)
{
  int ier = 0;
  int blocksize = 512;
  matrix_elementwise_multiply<<<(N - 1) / blocksize + 1, blocksize>>>(a, b, N);
  checkCudaErrors(hipDeviceSynchronize());
  return ier;
}

__global__ void matrix_elementwise_divide(CUCPX *a, PCS *b, int N)
{
  int idx;
  for (idx = threadIdx.x + blockIdx.x * blockDim.x; idx < N; idx += gridDim.x * blockDim.x)
  {
    a[idx].x = a[idx].x / b[idx];
    a[idx].y = a[idx].y / b[idx];
  }
}

int matrix_elementwise_divide_invoker(CUCPX *a, PCS *b, int N)
{
  int ier = 0;
  int blocksize = 512;
  matrix_elementwise_multiply<<<(N - 1) / blocksize + 1, blocksize>>>(a, b, N);
  checkCudaErrors(hipDeviceSynchronize());
  return ier;
}

void set_nhg_w(PCS S, PCS X, conv_opts spopts,
		     int &nf, PCS &h, PCS &gam)
/* sets nf, h (upsampled grid spacing), and gamma (x_j rescaling factor),
   for type 3 only.
   Inputs:
   X and S are the xj and sk interval half-widths respectively.
   opts and spopts are the NUFFT and spreader opts strucs, respectively.
   Outputs:
   nf is the size of upsampled grid for a given single dimension.
   h is the grid spacing = 2pi/nf
   gam is the x rescale factor, ie x'_j = x_j/gam  (modulo shifts).
   Barnett 2/13/17. Caught inf/nan 3/14/17. io int types changed 3/28/17
   New logic 6/12/17
*/
{
  int nss = spopts.kw + 1;      // since ns may be odd
  PCS Xsafe=X, Ssafe=S;              // may be tweaked locally
  if (X==0.0)                        // logic ensures XS>=1, handle X=0 a/o S=0
    if (S==0.0) {
      Xsafe=1.0;
      Ssafe=1.0;
    } else Xsafe = std::max(Xsafe, 1/S);
  else
    Ssafe = std::max(Ssafe, 1/X);
  // use the safe X and S...
  PCS nfd = 2.0*spopts.upsampfac*Ssafe*Xsafe/PI + nss;
  if (!isfinite(nfd)) nfd=0.0;                // use FLT to catch inf
  nf = (int)nfd;
  //printf("initial nf=%lld, ns=%d\n",*nf,spopts.nspread);
  // catch too small nf, and nan or +-inf, otherwise spread fails...
  if (nf<2*spopts.kw) nf=2*spopts.kw;
  h = 2*PI / nf;                            // upsampled grid spacing
  gam = (PCS)nf / (2.0*spopts.upsampfac*Ssafe);  // x scale fac to x'
}

void taylor_series_approx_factors(PCS *c0, PCS *c1, PCS *c2, PCS *c3, double beta, int N){
  for(int i=0; i<N; i++){
    double x = i / (double) N;
    c0[i] = exp(beta*sqrt(1-x*x));
    c1[i] = -beta*x*c0[i] / sqrt(1-x*x); //power
    c2[i] = - beta*(beta*x*x*pow((1-x*x),1.5) + x*x -1)*c0[i] / pow((1-x*x),1.5)/(x*x-1) /2; //some error here
    c3[i] = beta*x*(3*beta*pow((1-x*x),2.5)+beta*beta*pow(x,8)-3*beta*beta*pow(x,6)+(3*beta*beta-3)*pow(x,4)+(6-beta*beta)*x*x-3)*c0[i]/
            pow((1-x*x),2.5)/pow((x*x-1),2) /6;
  }
}

void taylor_series_approx_factors(PCS *c0, double beta, int N, int N_order, int func_type){
  for(int i=0; i<N; i++){
    double x = i / (double) N;
    c0[i*N_order] = exp(beta*(sqrt(1-x*x)-func_type));
    c0[i*N_order+1] =  -beta*x*c0[i*N_order] / sqrt(1-x*x);
    c0[i*N_order+2] = - beta*(beta*x*x*pow((1-x*x),1.5) + x*x -1)*c0[i*N_order] / pow((1-x*x),1.5)/(x*x-1) /2;
    c0[i*N_order+3] = beta*x*(3*beta*pow((1-x*x),2.5)+beta*beta*pow(x,8)-3*beta*beta*pow(x,6)+(3*beta*beta-3)*pow(x,4)+(6-beta*beta)*x*x-3)*c0[i*N_order]/
            pow((1-x*x),2.5)/pow((x*x-1),2) /6;
    c0[i*N_order+4] = beta*(6*beta*beta*pow(x,10)+(-24*beta*beta-12)*pow(x,8)+pow((1-x*x),3.5)*(beta*beta*beta*pow(x,6)-pow(beta,3)*pow(x,4)-12*beta*x*x-3*beta)+
                      (36*beta*beta+33)*pow(x,6)+(-24*beta*beta-27)*pow(x,4)+(6*beta*beta+3)*x*x+3)*c0[i*N_order] /
                      pow((1-x*x),3.5)/pow((x*x-1),3)/24;
  }
}