#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <thrust/complex.h>
#include <algorithm>
#include "utils.h"

int main(int argc, char* argv[]){

    PCS *arr, *d_arr;
    int n = 10;
    arr = (PCS *) malloc (sizeof(PCS)*n);
    for(int i=0; i<n; i++){
        arr[i] = randm11()*0.5*PI; //convert to int for checking
        printf("%.3g ", arr[i]);
    }
    printf("\n");
    checkCudaErrors(hipMalloc((void **)&d_arr, sizeof(PCS)*n));
    checkCudaErrors(hipMemcpy(d_arr, arr, sizeof(PCS)*n, hipMemcpyHostToDevice));

    /*-------------get_max_min test------------*/
    printf("Get max&min testing...\n");
    PCS max, min;
    get_max_min(max, min, d_arr, n);
    printf("max value is %.3g, min is %.3g\n", max, min);

    /*-------------prefix_scan test------------*/
    printf("Prefix scan testing...\n");
    prefix_scan(d_arr, d_arr, n, 0);
    checkCudaErrors(hipMemcpy(arr, d_arr, sizeof(PCS)*n, hipMemcpyDeviceToHost));
    for(int i=0; i<10; i++){
        printf("%.3g ", arr[i]);
    }
    printf("\n");

    free(arr);
    checkCudaErrors(hipFree(d_arr));

    /*-------------sparse histogram test--------------*/
    printf("Histogram testing...\n");
    n = 10;
    PCS *x; PCS *y ;PCS *z;
    PCS *d_x, *d_y, *d_z;
    PCS *d_x_out, *d_y_out, *d_z_out;

    CUCPX *d_c, *d_c_out;
    int *sortidx_bin, *histo_count;
    int *h_sortidx_bin, *h_histo_count;
    int2 *se_loc;
    int2 *h_se_loc;
    x = (PCS *)malloc(sizeof(PCS)*n);
    y = (PCS *)malloc(sizeof(PCS)*n);
    z = (PCS *)malloc(sizeof(PCS)*n);
    CPX *c;
    c = (CPX *)malloc(sizeof(CPX)*n);
    for(int i=0; i<n; i++){
        x[i] = randm11()*0.5*PI;
        y[i] = randm11()*0.5*PI;
        z[i] = randm11()*0.5*PI;
        c[i].real(i/double(n));
        c[i].real(i);
        printf("%.3g,%.3g ", x[i],y[i]);
    }
    printf("\n");

    int nf1 = n*2;
    int nf2 = n*2;
    int nf3 = 2;
    h_sortidx_bin = (int *)malloc(sizeof(int)*n);
    h_histo_count = (int *)malloc(sizeof(int)*(nf1*nf2+1));
    h_se_loc = (int2 *)malloc(sizeof(int2)*n);
    checkCudaErrors(hipMalloc((void **)&d_x, sizeof(PCS)*n));
    checkCudaErrors(hipMalloc((void **)&d_y, sizeof(PCS)*n));
    checkCudaErrors(hipMalloc((void **)&d_z,sizeof(PCS)*n));
    checkCudaErrors(hipMalloc((void **)&d_x_out, sizeof(PCS)*n));
    checkCudaErrors(hipMalloc((void **)&d_y_out, sizeof(PCS)*n));
    checkCudaErrors(hipMalloc((void **)&d_z_out,sizeof(PCS)*n));
    checkCudaErrors(hipMalloc((void **)&se_loc,sizeof(int2)*n));
    checkCudaErrors(hipMalloc((void **)&d_c,sizeof(CUCPX)*n));
    checkCudaErrors(hipMalloc((void **)&d_c_out,sizeof(CUCPX)*n));
    checkCudaErrors(hipMalloc((void**)&sortidx_bin,sizeof(int)*n));
    checkCudaErrors(hipMalloc((void**)&histo_count,sizeof(int)*(nf1*nf2+1)));
    
    checkCudaErrors(hipMemcpy(d_x,x,sizeof(PCS)*n,hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_y,y,sizeof(PCS)*n,hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_z,z,sizeof(PCS)*n,hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_c,c,sizeof(CPX)*n,hipMemcpyHostToDevice));


    checkCudaErrors(hipMemset(sortidx_bin,-1,sizeof(int)*n));
    checkCudaErrors(hipMemset(histo_count,0,sizeof(int)*(nf1*nf2+1)));
    int init_scan_value = 0;
    for(int i=0; i<nf3; i++){
        part_histogram_3d_sparse_invoker(d_x,d_y,d_z,sortidx_bin,histo_count,n,nf1,nf2,nf3,i,1);
        prefix_scan(histo_count,histo_count,nf1*nf2+1,0);
        part_mapping_based_gather_3d_invoker(d_x,d_y,d_z,d_c,d_x_out,d_y_out,d_z_out,d_c_out,sortidx_bin,histo_count,se_loc,n,nf1,nf2,nf3,i,init_scan_value,plan->copts.pirange);
        int last_value;
        checkCudaErrors(hipMemcpy(&last_value,histo_count+nf1*nf2,sizeof(int),hipMemcpyDeviceToHost));
        init_scan_value += last_value;
        checkCudaErrors(hipMemcpy(h_histo_count, histo_count, sizeof(int)*(nf1*nf2+1), hipMemcpyDeviceToHost));
        for(int i=0; i<nf1*nf2+1; i++){
            printf("%d ",h_histo_count[i]);
        }
        printf("\n");
        checkCudaErrors(hipMemset(histo_count,0,sizeof(int)*(nf1*nf2+1)));
    }
    checkCudaErrors(hipMemcpy(x,d_x_out,sizeof(PCS)*n,hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_se_loc,se_loc,sizeof(int2)*n,hipMemcpyDeviceToHost));
    for(int i=0; i<n; i++){
        printf("%lf,%d,%d ",x[i],h_se_loc[i].x,h_se_loc[i].y);
    }
    printf("\n");



    checkCudaErrors(hipMemcpy(h_sortidx_bin,sortidx_bin,sizeof(int)*n,hipMemcpyDeviceToHost));
    // printf("sortidx_bin\n");
    // for(int i=0; i<n; i++){
    //     printf("%d ",h_sortidx_bin[i]);
    // }
    // printf("\n");

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);
    hipFree(d_c);
    hipFree(d_x_out);
    hipFree(d_y_out);
    hipFree(d_z_out);
    hipFree(d_c_out);
    hipFree(sortidx_bin);
    hipFree(histo_count);
    hipFree(se_loc);
    free(x);
    free(y);
    free(z);
    free(c);
    free(h_se_loc);
    free(h_histo_count);
    free(h_sortidx_bin);
    /*-------------GPU info test------------*/
    GPU_info();
    return 0;
}